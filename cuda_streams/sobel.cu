#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "../stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../stb_image_write.h"

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <unistd.h>
#include <errno.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define BLOCK_SIZE 16
#define NUM_STREAMS 4  

__global__ void sobel_kernel(const uint8_t* input, uint8_t* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int Gx[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    };

    int Gy[3][3] = {
        {-1, -2, -1},
        { 0,  0,  0},
        { 1,  2,  1}
    };

    if (x >= width || y >= height) return;

    int sumX = 0, sumY = 0;
    for (int ky = -1; ky <= 1; ky++) {
        for (int kx = -1; kx <= 1; kx++) {
            int nx = x + kx;
            int ny = y + ky;
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                int idx = ny * width + nx;
                int pixel = input[idx];
                sumX += pixel * Gx[ky + 1][kx + 1];
                sumY += pixel * Gy[ky + 1][kx + 1];
            }
        }
    }

    int magnitude = sqrtf((float)(sumX * sumX + sumY * sumY));
    if (magnitude > 255) magnitude = 255;
    if (magnitude < 0) magnitude = 0;

    output[y * width + x] = (uint8_t)magnitude;
}

uint8_t* sobel_gpu_async(uint8_t* input, int width, int height, hipStream_t stream) {
    size_t img_size = width * height * sizeof(uint8_t);
    uint8_t *d_input = NULL, *d_output = NULL;
    uint8_t *output = (uint8_t*)malloc(img_size);
    if (!output) return NULL;

    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);

    hipMemcpyAsync(d_input, input, img_size, hipMemcpyHostToDevice, stream);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    sobel_kernel<<<gridDim, blockDim, 0, stream>>>(d_input, d_output, width, height);

    hipMemcpyAsync(output, d_output, img_size, hipMemcpyDeviceToHost, stream);

    // Free device memory after stream finishes
    hipStreamAddCallback(stream,
        [](hipStream_t stream, hipError_t status, void* userData) {
            uint8_t **devPtrs = (uint8_t**)userData;
            hipFree(devPtrs[0]);
            hipFree(devPtrs[1]);
            free(devPtrs);
        }, 
        malloc(sizeof(uint8_t*) * 2), 0);

    return output;
}

int has_image_extension(const char *filename) {
    const char *ext = strrchr(filename, '.');
    return ext && (strcmp(ext, ".jpg") == 0 || strcmp(ext, ".png") == 0);
}

typedef struct {
    char output_path[512];
    int width;
    int height;
    uint8_t *output_buffer;
} ImageResult;

int main(int argc, char *argv[]) {
    double total_start_time = omp_get_wtime();

    const char *input_folder = getenv("INPUT_DIR");
    if (argc > 1) input_folder = argv[1];
    if (!input_folder) {
        fprintf(stderr, "INPUT_DIR not set and no input folder given\n");
        return 1;
    }

    const char *output_folder = getenv("OUTPUT_DIR");
    if (argc > 2) output_folder = argv[2];
    if (!output_folder) {
        fprintf(stderr, "OUTPUT_DIR not set and no output folder given\n");
        return 1;
    }

    struct stat st = {0};
    if (stat(output_folder, &st) == -1) {
        if (mkdir(output_folder, 0755) != 0) {
            perror("Failed to create output directory");
            return 1;
        }
    }

    DIR *dir = opendir(input_folder);
    if (!dir) {
        perror("Failed to open input directory");
        return 1;
    }

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    struct dirent *entries[1024];
    int count = 0;
    struct dirent *entry;
    while ((entry = readdir(dir)) != NULL && count < 1024) {
        if (entry->d_type == DT_REG && has_image_extension(entry->d_name)) {
            entries[count] = (struct dirent*)malloc(sizeof(struct dirent));
            memcpy(entries[count], entry, sizeof(struct dirent));
            count++;
        }
    }
    closedir(dir);

    ImageResult results[1024] = {0};

    for (int i = 0; i < count; i++) {
        int stream_id = i % NUM_STREAMS;

        char input_path[512], output_path[512];
        snprintf(input_path, sizeof(input_path), "%s/%s", input_folder, entries[i]->d_name);
        snprintf(output_path, sizeof(output_path), "%s/sobel_%s", output_folder, entries[i]->d_name);

        int width, height, channels;
        unsigned char *img = stbi_load(input_path, &width, &height, &channels, 1);
        if (!img) {
            fprintf(stderr, "Failed to load %s\n", input_path);
            free(entries[i]);
            continue;
        }

        uint8_t *sobel = sobel_gpu_async(img, width, height, streams[stream_id]);
        stbi_image_free(img);

        strncpy(results[i].output_path, output_path, sizeof(results[i].output_path) - 1);
        results[i].width = width;
        results[i].height = height;
        results[i].output_buffer = sobel;

        free(entries[i]);
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }

    // Write images to disk
    for (int i = 0; i < count; i++) {
        if (results[i].output_buffer) {
            int stride_in_bytes = results[i].width * 1;
            if (!stbi_write_png(results[i].output_path, results[i].width, results[i].height, 1, results[i].output_buffer, stride_in_bytes)) {
                fprintf(stderr, "Failed to save %s\n", results[i].output_path);
            }
            free(results[i].output_buffer);
        }
    }

    // Destroy streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    double total_end_time = omp_get_wtime();
    printf("Total time: %f seconds\n", total_end_time - total_start_time);

    return 0;
}
